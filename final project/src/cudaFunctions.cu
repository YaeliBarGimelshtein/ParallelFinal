#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <iostream>
#include <string.h>
#include <limits.h>
#include "cudaHeader.h"


enum constants {WEIGHTS = 4,  RES = 3, A_ROWS = 9, A_COLS = 5, B_ROWS = 11 , B_COLS = 7};

__device__ __constant__ int a_rows = 9;
__device__ __constant__ int b_rows = 11;
__device__ __constant__ int a_cols = 5;
__device__ __constant__ int b_cols = 7;
__device__ __constant__ int res_components = 3;

__device__ __constant__ char levelA[A_ROWS][A_COLS] = {"NDEQ", "MILV", "FYW", "NEQK", "QHRK", "HY", "STA", "NHQK", "MILF"};
__device__ __constant__ char levelB[B_ROWS][B_COLS] = {"SAG", "ATV", "CSA", "SGND", "STPA", "STNK", "NEQHRK", "NDEQHK", "SNDEQK", "HFY", "FVLIM"};

__device__ int cuda_strlen(char* seq)
{
    int counter = 0;
    while(*seq != '\0')
    {
        counter++;
        seq++;
    }

    return counter;
}

__device__ int check_level_A_cuda(char from_seq1, char from_seq2)
{
    char from_seq1_new = from_seq1;
    char from_seq2_new = from_seq2;

    //UPPER CASE
    if(from_seq1 >= 'a' && from_seq1 <='z')
        from_seq1_new = from_seq1 - 32;
    
    if(from_seq2 >= 'a' && from_seq2 <='z')
        from_seq2_new = from_seq2 - 32;

    int counter_1 = 0, counter_2 = 0;
    for (int i = 0; i < a_rows; i++)
    {
        for (int j = 0; j < cuda_strlen(levelA[i]); j++)
        {
            if(from_seq1_new == levelA[i][j])
                counter_1++;
            
            if(from_seq2_new == levelA[i][j])
                counter_2++;
        }

        if(counter_1 > 0 && counter_2 > 0)
            return 1;

        counter_1 = 0;
        counter_2 = 0;
        
    }
    return 0;
}



__device__ int check_level_B_cuda(char from_seq1, char from_seq2)
{
    char from_seq1_new = from_seq1;
    char from_seq2_new = from_seq2;

    //UPPER CASE
    if(from_seq1 >= 'a' && from_seq1 <='z')
        from_seq1_new = from_seq1 - 32;
    
    if(from_seq2 >= 'a' && from_seq2 <='z')
        from_seq2_new = from_seq2 - 32;

    int counter_1 = 0, counter_2 = 0;
    for (int i = 0; i < b_rows; i++)
    {
        for (int j = 0; j < cuda_strlen(levelB[i]); j++)
        {
            if(from_seq1_new == levelB[i][j])
                counter_1++;
            
            if(from_seq2_new == levelB[i][j])
                counter_2++;
        }

        if(counter_1 > 0 && counter_2 > 0)
            return 1;
        
        counter_1 = 0;
        counter_2 = 0;
        
    }
    return 0;
}



__global__ void return_score_offset_mutant(char* seq1, char* seq2 , int* lenght_seq2 , int* weights, int* start, int* finish, int* res)
{
    int possible_mutants = *lenght_seq2;
    int curr_score = 0;
    int index = 0;

    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    int offset =  thread_id + *start;
    int thread_index_for_res_array = thread_id * res_components; 

    int dollars = 0, precent = 1, hashes = 2, spaces = 3; 

    if(offset <= *finish)
    {
        for (int mutant = 0; mutant < possible_mutants; mutant++) //all mutants
        {
            index = offset;
            for (int index_seq2 = 0; index_seq2 < *lenght_seq2; index_seq2++)
            {
                //MUTATION POSSIBLE ONLY IF NOT THE FINAL OFFSET
                if(offset < *finish)
                {
                    if (index_seq2 == mutant && mutant != 0)
                        index++;
                }
    
                
                if (seq1[index] == seq2[index_seq2])
                    curr_score += weights[dollars];
                
                else if (check_level_A_cuda(seq1[index], seq2[index_seq2]))
                    curr_score -= weights[precent];
                
                else if (check_level_B_cuda(seq1[index], seq2[index_seq2]))
                    curr_score -= weights[hashes];
                
                else
                    curr_score -= weights[spaces];

                index++;
            }
            
            if(curr_score > res[thread_index_for_res_array])
            {
                res[thread_index_for_res_array] = curr_score;
                res[thread_index_for_res_array + 1] = offset;
                res[thread_index_for_res_array + 2] = mutant;
            }
            curr_score = 0;
        }
    } 
}

int* return_cuda_score_offset_mutant(char* seq1, char* seq2 , int* weights, int offset_start, int offset_finish)
{
    //CREATE THE ARRAY THAT WILL RETURN THE BEST RESULT
    int* result = (int*)calloc(sizeof(int), RES);

    //DATA NEEDED
    int offset_size = offset_finish - offset_start + 1; //--> num of cuda threads
    int num_of_blocks = (offset_size / NUM_THREADS_PER_BLOCK);
    if (offset_size % NUM_THREADS_PER_BLOCK != 0)
        num_of_blocks ++;
    
    int lenght_of_res = (RES) * offset_size; //for each offset the gpu finds the best mutant

    //ALLOCATE DATA TO CUDA MEMORY
    char* cuda_seq1, *cuda_seq2;
    int* cuda_weights, *cuda_offset_start, *cuda_offset_finish ,*cuda_seq2_lenght;
    int* cuda_res, *res = (int*)calloc(sizeof(int) , lenght_of_res);
    int seq2_lenght = (strlen(seq2));

    for (int i = 0; i < lenght_of_res; i++) // get the array ready
    {
        res[i] = INT_MIN;
    }

        //sizes to allocate
    int size_for_cuda_seq1 = sizeof(char) * (strlen(seq1));
    int size_for_cuda_seq2 = sizeof(char) * seq2_lenght;
    int size_for_cuda_weights = sizeof(int) * (WEIGHTS);
    int size_for_cuda_int = sizeof(int);
    int size_for_cuda_res = sizeof(int) * lenght_of_res;

        //allocate
    hipMalloc((void**)&cuda_seq1, size_for_cuda_seq1);
    hipMalloc((void**)&cuda_seq2, size_for_cuda_seq2);
    hipMalloc((void**)&cuda_weights, size_for_cuda_weights);
    hipMalloc((void**)&cuda_offset_start, size_for_cuda_int);
    hipMalloc((void**)&cuda_offset_finish, size_for_cuda_int);
    hipMalloc((void**)&cuda_res, size_for_cuda_res);
    hipMalloc((void**)&cuda_seq2_lenght, size_for_cuda_int);



    //COPY INPUT INTO DEVICE
    hipMemcpy(cuda_seq1, seq1, size_for_cuda_seq1, hipMemcpyHostToDevice);
    hipMemcpy(cuda_seq2, seq2, size_for_cuda_seq2, hipMemcpyHostToDevice);
    hipMemcpy(cuda_weights, weights, size_for_cuda_weights, hipMemcpyHostToDevice);
    hipMemcpy(cuda_offset_start, &offset_start, size_for_cuda_int, hipMemcpyHostToDevice);
    hipMemcpy(cuda_offset_finish, &offset_finish, size_for_cuda_int, hipMemcpyHostToDevice);
    hipMemcpy(cuda_seq2_lenght, &seq2_lenght, size_for_cuda_int, hipMemcpyHostToDevice);
    hipMemcpy(cuda_res, res, size_for_cuda_res, hipMemcpyHostToDevice);
    
    
    //LUNCH KERNEL
    return_score_offset_mutant<<<num_of_blocks, NUM_THREADS_PER_BLOCK>>>(cuda_seq1, cuda_seq2, cuda_seq2_lenght, cuda_weights, cuda_offset_start, cuda_offset_finish ,cuda_res);

    //COPY RESULT BACK TO HOST
    hipMemcpy(res, cuda_res, size_for_cuda_res, hipMemcpyDeviceToHost);
    
    //GET THE BIGGEST SCORE
    result[0] = INT_MIN; 

    for (int i = 0; i < lenght_of_res; i += 3)
    {
        if(result[0] < res[i])
        {
            result[0] = res[i];
            result[1] = res[i+1];
            result[2] = res[i+2];
        }
    }
    
    //FREE
    hipFree(cuda_seq1);
    hipFree(cuda_seq2);
    hipFree(cuda_weights);
    hipFree(cuda_offset_start);
    hipFree(cuda_offset_finish);
    hipFree(cuda_res);
    hipFree(cuda_seq2_lenght);

    return result;
}
